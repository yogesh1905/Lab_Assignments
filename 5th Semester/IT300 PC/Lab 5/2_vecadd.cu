#define N 5000
#define T 1024 // max threads per block


#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void vecAdd (int *a, int *b, int *c);
void printArray(int a[], int b[], int c[]);

int main() {
	int a[N], b[N], c[N];
	int *dev_a, *dev_b, *dev_c;

	// initialize a and b with real values (NOT SHOWN)
	int size = N * sizeof(int);
	for (int i = 0; i < N; i++) {
		a[i] = i;
		b[i] = i/2;
	}

	hipMalloc((void**)&dev_a, size);
	hipMalloc((void**)&dev_b, size);
	hipMalloc((void**)&dev_c, size);

	hipMemcpy(dev_a, a, size,hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, size,hipMemcpyHostToDevice);

	vecAdd<<<(int)ceil(N/T),T>>>(dev_a,dev_b,dev_c);

	hipMemcpy(c, dev_c, size,hipMemcpyDeviceToHost);

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	
	printArray(a,b,c);

	exit (0);
}

__global__ void vecAdd (int *a, int *b, int *c) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < N) {
		c[i] = a[i] + b[i];
	}
}

void printArray(int a[], int b[], int c[]) {

	printf("Array a:\n");
	for(int i = 0; i < N; i++){
		printf("%d ", a[i]);
    }
    printf("\n\nArray b:\n");
    for(int i = 0; i < N; i++){
		printf("%d ", b[i]);
    }
    printf("\n\nArray c:\n");
    for(int i = 0; i < N; i++){
		printf("%d ", c[i]);
	}
	printf("\n");
}
